#include "hip/hip_runtime.h"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>

#include "Timer.hpp"
#include <iostream>
#include <string>

#include "hip/hip_runtime_api.h"

template<typename T>
struct add {
  __host__ __device__ T operator()(const T& x, const T& y) const { return 3.14159 * x + y; }
};

size_t frob(size_t size) {
  size_t r = 0;
  while(size >>= 1) {
    ++r;
  }
  return 1 << r;
}

size_t find_10ms_size () {
  size_t size = 4;
  Timer t0;

  for (; size < 4UL * 1024 * 1024 * 1024; size *= 2) {
    size_t N = size;
    thrust::device_vector<float> X(N), Y(N);
    t0.start();
    checkCudaErrors(hipDeviceSynchronize());
    thrust::transform(X.begin(), X.end(), Y.begin(), Y.begin(), [=]__host__ __device__(const float x, const float y) { return 3.14159*x + y; });
    checkCudaErrors(hipDeviceSynchronize());
    t0.stop();
    if (t0.elapsed() >= 10.0) {
      break;
    }
  }

  double rat = std::max<double>(1.0, std::ceil(t0.elapsed()/10.0));
  size /= static_cast<size_t>(rat);

  return frob(size);
}


int main(int argc, char* argv[]) {

  size_t N = 1 << 16;

  if (argc >= 2) {
    size_t nn = std::stol(argv[1]);
    N = frob(1 << nn);
  }

  DEF_TIMER(host_vector); START_TIMER(host_vector);
  thrust::host_vector<float>   x(N), y(N);
  STOP_TIMER(host_vector);
  DEF_TIMER(device_vector); START_TIMER(device_vector);
  thrust::device_vector<float> X(N), Y(N);
  STOP_TIMER(device_vector);

  for (int i = 0; i < N; i++)
    y[i] = 2.0 * (x[i] = 1.0f);
  
  DEF_TIMER(copy);    START_TIMER(copy);
  thrust::copy(x.begin(), x.end(), X.begin());
  thrust::copy(y.begin(), y.end(), Y.begin());
  STOP_TIMER(copy);

  size_t num_trips = 50.0 * find_10ms_size() / static_cast<double>(N);
  num_trips = std::max<size_t>(num_trips, 5);
  
  double a = 3.14159;

  DEF_TIMER(thrust_call); START_TIMER(thrust_call);
  hipDeviceSynchronize();
  for (size_t i = 0; i < num_trips; ++i) {
    thrust::transform(X.begin(), X.end(), Y.begin(), Y.begin(), [=]__host__ __device__(const float x, const float y) { return a*x + y; });
  
    hipDeviceSynchronize();
  }
  STOP_TIMER(thrust_call);

  double gflops = 2.0 * num_trips * N / 1.e9;
  double gflops_sec = gflops / (t_thrust_call.elapsed() * 1.e-3);
  std::cout << "# gflops / sec [ madd ]: " << gflops_sec << std::endl;

  DEF_TIMER(copy_back); START_TIMER(copy_back);
  thrust::copy(Y.begin(), Y.end(), y.begin());
  STOP_TIMER(copy_back);

  return 0;
}
