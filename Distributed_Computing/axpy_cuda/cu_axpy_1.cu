#include "hip/hip_runtime.h"


#include "Timer.hpp"
#include <iostream>
#include <string>

#include "hip/hip_runtime_api.h"

__global__ void madd(int n, float a, float* x, float* y) {
  for (int i = 0; i < n; i++)
    y[i] = a * x[i] + y[i];
}

size_t frob(size_t size) {
  size_t r = 0;
  while(size >>= 1) {
    ++r;
  }
  return 1 << r;
}

size_t find_10ms_size () {
  size_t size = 4;
  Timer t0;

  for (; size < 4UL * 1024 * 1024 * 1024; size *= 2) {
    float *x, *y;
    size_t N = size;

    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));
    t0.start();
    checkCudaErrors(hipDeviceSynchronize());
    madd<<<1, 1>>>(N, 3.14159, x, y);
    checkCudaErrors(hipDeviceSynchronize());
    t0.stop();
    hipFree(x);
    hipFree(y);
    if (t0.elapsed() >= 10.0) {
      break;
    }
  }

  double rat = std::max<double>(1.0, std::ceil(t0.elapsed()/10.0));
  size /= static_cast<size_t>(rat);

  return frob(size);
}


int main(int argc, char* argv[]) {

  size_t N = 1 << 16;

  if (argc >= 2) {
    size_t nn = std::stol(argv[1]);
    N = frob(1 << nn);
  }

  float *x = nullptr, *y = nullptr;

  DEF_TIMER(cuda_malloc); START_TIMER(cuda_malloc);
  hipMallocManaged(&x, N * sizeof(float));
  hipMallocManaged(&y, N * sizeof(float));
  STOP_TIMER(cuda_malloc);

  size_t num_trips = 50.0 * find_10ms_size() / static_cast<double>(N);
  num_trips = std::max<size_t>(num_trips, 5);
  
  for (int i = 0; i < N; i++)
    y[i] = 2.0 * (x[i] = 1.0f);
  
  DEF_TIMER(cuda_call);
  START_TIMER(cuda_call);
  hipDeviceSynchronize();
  for (size_t i = 0; i < num_trips; ++i) {
    madd<<<1, 1>>>(N, 3.1459, x, y);
    hipDeviceSynchronize();
  }
  STOP_TIMER(cuda_call);

  double gflops = 2.0 * num_trips * N / 1.e9;
  double gflops_sec = gflops / (t_cuda_call.elapsed() * 1.e-3);
  std::cout << "# gflops / sec [ madd ]: " << gflops_sec << std::endl;

  DEF_TIMER(cuda_free); START_TIMER(cuda_free);
  hipFree(x);
  hipFree(y);
  STOP_TIMER(cuda_free);

  return 0;
}
