#include "hip/hip_runtime.h"
//
// This file is part of the course materials for AMATH483/583 at the University of Washington,
// Spring 2020
//
// Licensed under Creative Commons Attribution-NonCommercial-ShareAlike 4.0 International License
// https://creativecommons.org/licenses/by-nc-sa/4.0/
//
// Author: Andrew Lumsdaine
//

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>

#include <cmath>
#include <cstddef>
#include <iomanip>
#include <iostream>
#include <string>
#include <vector>

#include "Timer.hpp"
#include "Vector.hpp"
#include "amath583.hpp"

template<typename T>
T norm_thrust(const thrust::device_vector<T>& x) {
  // Write me
  T sum = 0;
  return std::sqrt(sum);
}


void header(const std::string& msg = "") {
  auto& os_ = std::cout;
  if (msg != "") {
    os_ << std::setw(12) << std::left << msg << std::endl;
  }
  os_ << std::setw(12) << std::right << "N";
  os_ << std::setw(12) << std::right << "Sequential";

  os_ << std::setw(12) << std::right << "First";
  os_ << std::setw(12) << std::right << "Second";

  os_ << std::setw(12) << std::right << "First";
  os_ << std::setw(12) << std::right << "Second";

  os_ << std::endl;
}

double Gflops_sec(size_t nnz, size_t trials, double msec) {
  double Gflops = static_cast<double>(trials) * (2.0 * nnz) / 1.e9;
  double sec    = msec / 1.e3;
  if (sec == 0) {
    return 0;
  }
  return Gflops / sec;
}

void log(size_t nnz, size_t ntrials, const std::vector<double>& ms_times, const std::vector<double>& norms) {
  auto& os_ = std::cout;
  os_ << std::setw(12) << std::right << nnz;

  for (size_t i = 0; i < ms_times.size(); ++i) {
    os_ << std::setw(12) << std::right << Gflops_sec(nnz, ntrials, ms_times[i]);
  }
  for (size_t i = 1; i < ms_times.size(); ++i) {
    os_ << std::setw(14) << std::right << std::abs(norms[i] - norms[0]) / norms[0];
  }
  os_ << std::endl;
}

size_t num_trials(size_t nnz) {
  // trials*nnz/1G = 1S => trials = 1G/nnz

  double N_1k = std::ceil(1E9 / static_cast<double>(nnz));

  return 5 + static_cast<size_t>(N_1k);
}

template <class T, typename Function>
void run_cu(Function&& f, size_t N_min, size_t N_max) {
  header(sizeof(T) == 4 ? "\nFloat" : "\nDouble");
  Timer t;

  for (size_t size = N_min; size <= N_max; size *= 2) {
    std::vector<double> ms_times;
    std::vector<double> norms;

    Vector            x(size);
      
    randomize(x);

    double norm0 = two_norm(x);
    double norm1 = 0.0;

    size_t ntrials = num_trials(size);

    t.start();
    for (size_t i = 0; i < ntrials; ++i) {
      norm0 = two_norm(x);
    }
    t.stop();
    ms_times.push_back(t.elapsed());
    norms.push_back(norm0);

    thrust::device_vector<T> X(size);
    thrust::copy(&x(0), &x(0)+x.num_rows(), X.begin());

    for (size_t trip = 0; trip < 2; ++trip) {

      t.start();
      hipDeviceSynchronize();
      for (size_t i = 0; i < ntrials; ++i) {
        norm1 = f(X);
        hipDeviceSynchronize();
      }
      t.stop();
      ms_times.push_back(t.elapsed());
      norms.push_back(norm1);
    }

    log(size, ntrials, ms_times, norms);
  }
}




int main(int argc, char* argv[]) {
  size_t N_min = 1024 * 1024;
  size_t N_max = 128 * 1024 * 1024;

  if (argc >= 2) {
    N_min = std::stol(argv[1]);
  }
  if (argc >= 3) {
    N_max = std::stol(argv[2]);
  }

  run_cu<float>(norm_thrust<float>, N_min, N_max);
  run_cu<double>(norm_thrust<double>, N_min, N_max);

  return 0;
}
